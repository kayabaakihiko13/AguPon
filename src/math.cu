#include "hip/hip_runtime.h"
// #include <math.h>
#include "include/math.hh"
#include <hip/hip_runtime.h>
__global__ void kernelAddVectors(float* a, float* b, float* c, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        c[index] = a[index] + b[index];
    }
}

void addVectors(float* a, float* b, float* c, int size) {
    // Allocate device memory
    float *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, size * sizeof(float));
    hipMalloc((void**)&d_b, size * sizeof(float));
    hipMalloc((void**)&d_c, size * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_a, a, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size * sizeof(float), hipMemcpyHostToDevice);

    // Define block size and grid size
    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;

    // Launch kernel
    kernelAddVectors<<<numBlocks, blockSize>>>(d_a, d_b, d_c, size);

    // Copy result from device to host
    hipMemcpy(c, d_c, size * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
__global__
void MutVectorGPU(int* a,int*b,int*c,int N){
    // using kernel
    int tid = threadIdx.x * blockDim.x * blockIdx.x;
    c[tid] = a[tid] * b[tid];
}